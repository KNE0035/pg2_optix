#include "hip/hip_runtime.h"
#include "optixtutorial.h"

struct TriangleAttributes
{
	optix::float3 normal;
	optix::float2 texcoord;
};

rtBuffer<optix::float3, 1> normal_buffer;
rtBuffer<optix::float2, 1> texcoord_buffer;
rtBuffer<optix::uchar4, 2> output_buffer;

rtDeclareVariable( optix::float3, diffuse, , "diffuse" );
rtDeclareVariable(int, tex_diffuse_id, , "diffuse texture id");

rtDeclareVariable( rtObject, top_object, , );
rtDeclareVariable( uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable( uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable( PerRayData_radiance, ray_data, rtPayload, );
rtDeclareVariable( PerRayData_shadow, shadow_ray_data, rtPayload, );
rtDeclareVariable( float2, barycentrics, attribute rtTriangleBarycentrics, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(TriangleAttributes, attribs, attribute attributes, "Triangle attributes");
rtDeclareVariable(optix::float3, view_from, , );
rtDeclareVariable(optix::Matrix3x3, M_c_w, , "camera to worldspace transformation matrix" );
rtDeclareVariable(float, focal_length, , "focal length in pixels" );


RT_PROGRAM void attribute_program( void )
{
	const optix::float2 barycentrics = rtGetTriangleBarycentrics();
	const unsigned int index = rtGetPrimitiveIndex();
	const optix::float3 n0 = normal_buffer[index * 3 + 0];
	const optix::float3 n1 = normal_buffer[index * 3 + 1];
	const optix::float3 n2 = normal_buffer[index * 3 + 2];

	const optix::float2 t0 = texcoord_buffer[index * 3 + 0];
	const optix::float2 t1 = texcoord_buffer[index * 3 + 1];
	const optix::float2 t2 = texcoord_buffer[index * 3 + 2];

	attribs.normal = optix::normalize(n1 * barycentrics.x + n2 * barycentrics.y + n0 * (1.0f - barycentrics.x - barycentrics.y));
	attribs.texcoord = t1 * barycentrics.x + t2 * barycentrics.y + t0 * (1.0f - barycentrics.x - barycentrics.y);

	if (optix::dot(ray.direction, attribs.normal) > 0) {
		attribs.normal *= -1;
	}
}

RT_PROGRAM void primary_ray( void )
{
	PerRayData_radiance prd;
	hiprandState_t state;
	prd.state = &state;
	hiprand_init(launch_index.x + launch_dim.x * launch_index.y, 0, 0, prd.state);

	int ANTI_ALIASING_SAMPLES = 8;
	int NO_SAMPLES = 30;

	optix::float3 resultColor = optix::make_float3(0.0f, 0.0f, 0.0f);
	for (int i = 0; i < ANTI_ALIASING_SAMPLES; i++)
	{
		float randomX = hiprand_uniform(prd.state);
		float randomY = hiprand_uniform(prd.state);

		const optix::float3 d_c = make_float3(launch_index.x - launch_dim.x * 0.5f + randomX, 
											  output_buffer.size().y * 0.5f - launch_index.y + randomY, 
											  -focal_length);

		const optix::float3 d_w = optix::normalize(M_c_w * d_c);
		optix::Ray ray(view_from, d_w, 0, 0.01f);

		for (int j = 0; j < NO_SAMPLES; j++) {
			rtTrace(top_object, ray, prd);
			resultColor += prd.result;
		}
	}
	resultColor /= ANTI_ALIASING_SAMPLES;
	output_buffer[launch_index] = optix::make_uchar4(resultColor.x*255.0f, resultColor.y*255.0f, resultColor.z*255.0f, 255 );
}

RT_PROGRAM void closest_hit_normal_shader( void )
{
	optix::float3 normal = attribs.normal;
	ray_data.result = optix::make_float3((normal.x + 1) / 2, (normal.y + 1) / 2, (normal.z + 1) / 2);
}

enum class Shader : char { NORMAL = 1, LAMBERT = 2, PHONG = 3, GLASS = 4, PBR = 5, MIRROR = 6, TS = 7, CT = 8 };

RT_PROGRAM void closest_hit_lambert_shader(void)
{
	ray_data.depth++;
	//printf("%d \n", ray_data.depth);
	//printf("%d \n", ray.ray_type);
	optix::float3 lightPossition = optix::make_float3(7, 0, 120);
	
	optix::float3 intersectionPoint = optix::make_float3(ray.origin.x + ray.tmax * ray.direction.x, 
														ray.origin.y + ray.tmax * ray.direction.y,
														ray.origin.z + ray.tmax * ray.direction.z);

	optix::float3 vectorToLight = optix::normalize(lightPossition - intersectionPoint);
	optix::float3 normal = attribs.normal;

	float normalLigthScalarProduct = optix::dot(vectorToLight, normal);
	
	float pdf = 0;
	optix::float3 omegai = sampleHemisphere(normal, ray_data.state, pdf);
	
	optix::Ray ray(intersectionPoint, omegai, 1, 0.01f);
	PerRayData_shadow shadow_ray;
	shadow_ray.visible.x = 1;
	rtTrace(top_object, ray, shadow_ray);

	optix::float3 color;

	if (tex_diffuse_id != -1) {
		const optix::float4 value = optix::rtTex2D<optix::float4>(tex_diffuse_id, attribs.texcoord.x, 1 - attribs.texcoord.y);
		color = optix::make_float3(value.x, value.y, value.z);
	}
	else {
		color = optix::make_float3(diffuse.x, diffuse.y, diffuse.z);
	}

	ray_data.result = color * (normalLigthScalarProduct * optix::dot(normal, omegai) * shadow_ray.visible.x * (1 / HIP_PI_F * pdf));
}

RT_PROGRAM void closest_hit_phong_shader(void)
{
}

RT_PROGRAM void closest_hit_glass_shader(void)
{
}

RT_PROGRAM void closest_hit_pbr_shader(void)
{
}

RT_PROGRAM void closest_hit_mirror_shader(void)
{
}

RT_PROGRAM void any_hit(void)
{
	//if (diffuse.x == 1.0) {
		shadow_ray_data.visible.x = 0;
	//}
	//else {
	//	shadow_ray_data.visible.x = 0;
	//}
	rtTerminateRay();
}


/* may access variables declared with the rtPayload semantic in the same way as closest-hit and any-hit programs */
RT_PROGRAM void miss_program( void )
{
	ray_data.result = optix::make_float3( 0.0f, 0.0f, 0.0f );
}

RT_PROGRAM void exception( void )
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Exception 0x%X at (%d, %d)\n", code, launch_index.x, launch_index.y );
	rtPrintExceptionDetails();
	output_buffer[launch_index] = uchar4{ 255, 0, 255, 0 };
}


__device__ optix::float3 sampleHemisphere(optix::float3 normal, hiprandState_t* state, float& pdf) {
	float randomU = hiprand_uniform(state);	
	float randomV = hiprand_uniform(state);

	float x = 2 * cosf(2 * HIP_PI_F * randomU) * sqrtf(randomV * (1 - randomV));
	float y = 2 * sinf(2 * HIP_PI_F * randomU) * sqrtf(randomV * (1 - randomV));
	float z = 1 - 2 * randomV;

	optix::float3 omegai = optix::make_float3(x, y , z);

	if (optix::dot(normal, omegai) < 0) 
	{
		omegai *= -1;
	}

	pdf = 1 / (2 * HIP_PI_F);
	return omegai;
}