#include "hip/hip_runtime.h"
#include "optixtutorial.h"

struct TriangleAttributes
{
	optix::float3 normal;
	optix::float2 texcoord;
};

rtBuffer<optix::float3, 1> normal_buffer;
rtBuffer<optix::float2, 1> texcoord_buffer;
rtBuffer<optix::uchar4, 2> output_buffer;

rtDeclareVariable( rtObject, top_object, , );
rtDeclareVariable( uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable( uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable( PerRayData_radiance, ray_data, rtPayload, );
rtDeclareVariable( float2, barycentrics, attribute rtTriangleBarycentrics, );
//rtDeclareVariable(TriangleAttributes, attribs, attribute attributes, "Triangle attributes");

rtDeclareVariable(optix::float3, view_from, , );
rtDeclareVariable(optix::Matrix3x3, M_c_w, , "camera to worldspace transformation matrix" );
rtDeclareVariable(float, focal_length, , "focal length in pixels" );

/*
RT_PROGRAM void attribute_program( void )
{
	const unsigned int index = rtGetPrimitiveIndex();
	const optix::float3 n0 = normal_buffer[index * 3 + 0];
	const optix::float3 n1 = normal_buffer[index * 3 + 1];
	const optix::float3 n2 = normal_buffer[index * 3 + 2];
	optix::float3 normal = optix::normalize(n1 * barycentrics.x + n2 * barycentrics.y + n0 * (1.0f - barycentrics.x - barycentrics.y));

	//attribs.normal = optix::normalize(n1 * barycentrics.x + n2 * barycentrics.y + n0 * (1.0f - barycentrics.x - barycentrics.y));
}*/

RT_PROGRAM void primary_ray( void )
{
	/*if ( launch_index.x == 630)
	{
		//rtPrintf( "(%u, %u)\n", launch_dim.x, launch_dim.y );
		rtPrintf("(%u, %u)\n", output_buffer.size().x, output_buffer.size().y);
		output_buffer[launch_index] = optix::make_uchar4(255.0f, 255.0f, 255.0f, 255.0f);
		return;
	}*/

	const optix::float3 d_c = make_float3(launch_index.x -
		launch_dim.x * 0.5f, output_buffer.size().y * 0.5f -
		launch_index.y, -focal_length);
	const optix::float3 d_w = optix::normalize(M_c_w * d_c);
	optix::Ray ray(view_from, d_w, 0, 0.01f);

	PerRayData_radiance prd;
	rtTrace( top_object, ray, prd );

	// access to buffers within OptiX programs uses a simple array syntax	
	output_buffer[launch_index] = optix::make_uchar4( prd.result.x*255.0f, prd.result.y*255.0f, prd.result.z*255.0f, 255 );
}

RT_PROGRAM void closest_hit( void )
{
	const unsigned int index = rtGetPrimitiveIndex();
	const optix::float3 n0 = normal_buffer[index * 3 + 0];
	const optix::float3 n1 = normal_buffer[index * 3 + 1];
	const optix::float3 n2 = normal_buffer[index * 3 + 2];
	optix::float3 normal = optix::normalize(n1 * barycentrics.x + n2 * barycentrics.y + n0 * (1.0f - barycentrics.x - barycentrics.y));

	ray_data.result = optix::make_float3((normal.x + 1) / 2, (normal.y + 1) / 2, (normal.z + 1) / 2);
}

/* may access variables declared with the rtPayload semantic in the same way as closest-hit and any-hit programs */
RT_PROGRAM void miss_program( void )
{
	ray_data.result = optix::make_float3( 0.0f, 0.0f, 0.0f );
}

RT_PROGRAM void exception( void )
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Exception 0x%X at (%d, %d)\n", code, launch_index.x, launch_index.y );
	rtPrintExceptionDetails();
	output_buffer[launch_index] = uchar4{ 255, 0, 255, 0 };
}
